#include "hip/hip_runtime.h"
/**
 *  Copyright 2009 Universita' degli Studi di Pavia
 *  Laboratorio di Visione Artificiale
 *  http://vision.unipv.it
 * 
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.

 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.

 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */


// This enables CUDA-specific debug checks
// #define _DEBUG


#ifndef FIND_WINNERS_KERNEL
#define FIND_WINNERS_KERNEL
#endif

#include <float.h>
#include <math.h>
#include <cutil.h>

#include "landmarks.h"
#include "findWinnersKernel.h"


__constant__ int d_dimension = 0;
__constant__ int d_max_landmarks = 0;
__constant__ int d_max_samples = 0;

__constant__ float4 *d_landmarks = 0;
__constant__ float4 *d_samples = 0;
__constant__ int2 *d_indices = 0;
__constant__ float2 *d_distances = 0;


/* This version is for dimensions up to 4 */
__device__ void findWinnerTile4(int tile, float4 sample, int2 *bestMatching, float2 *bestDistance) {

	extern __shared__ float4 sharedLandmarks[];

	// fprintf(stderr, "findWinnersCuda.cu: findWinnerTile4 %d tile %d\n", blockIdx.x * blockDim.x + threadIdx.x, tile);

	if (tile + threadIdx.x < d_max_landmarks) {
		// Load landmark
		sharedLandmarks[threadIdx.x] = d_landmarks[tile + threadIdx.x];
	}
	else {
		landmarks_markInactiveCell(&sharedLandmarks[threadIdx.x].x);
	}

	__syncthreads();

	float tmp;
	float distance;

	for (int i = 0; i < blockDim.x; ++i) {
		tmp = sharedLandmarks[i].x;
		if (landmarks_isActiveCell(&tmp)) {
			tmp = sample.x - tmp;
			distance = tmp * tmp;
			tmp = sample.y - sharedLandmarks[i].y;
			distance += tmp * tmp;
			tmp = sample.z - sharedLandmarks[i].z;
			distance += tmp * tmp;
			tmp = sample.w - sharedLandmarks[i].w;
			distance += tmp * tmp;

			if (distance < (*bestDistance).x) {
				(*bestMatching).y = (*bestMatching).x;
				(*bestDistance).y = (*bestDistance).x;

				(*bestMatching).x = tile + i;
				(*bestDistance).x = distance;

				/*
				fprintf(stderr, "findWinnersCuda.cu: findWinnerTile4 %d tile %d bestMatching %d [%f,%f,%f,%f]->[%f,%f,%f,%f] distance %f\n", 
					blockIdx.x * blockDim.x + threadIdx.x, tile, (*bestMatching).x,
					sample.x, sample.y, sample.z, sample.w,
					sharedLandmarks[i].x, sharedLandmarks[i].y, sharedLandmarks[i].z, sharedLandmarks[i].w, tmp);
				*/
			} 
			else if (distance < (*bestDistance).y) {
				(*bestMatching).y = tile + i;
				(*bestDistance).y = distance;

				/*
				fprintf(stderr, "findWinnersCuda.cu: findWinnerTile4 %d tile %d secondBestMatching %d [%f,%f,%f,%f]->[%f,%f,%f,%f] distance %f\n", 
					blockIdx.x * blockDim.x + threadIdx.x, tile, (*bestMatching).y,
					sample.x, sample.y, sample.z, sample.w,
					sharedLandmarks[i].x, sharedLandmarks[i].y, sharedLandmarks[i].z, sharedLandmarks[i].w, tmp);
				*/
			}
		}
	}
}


/* This version is for dimensions up to 4 */
__global__ void findWinner4() {

	float4 sample;
	int2 bestMatching = {-1, -1};
	float2 bestDistance = {FLT_MAX, FLT_MAX};

	// fprintf(stderr, "findWinnersCuda.cu: findWinner4 %d load\n", blockIdx.x * blockDim.x + threadIdx.x);

	if (blockIdx.x * blockDim.x + threadIdx.x < d_max_samples) {
		// Load sample
		sample = d_samples[blockIdx.x * blockDim.x + threadIdx.x];

		// fprintf(stderr, "findWinnersCuda.cu: findWinner4 %d browse tiles\n", blockIdx.x * blockDim.x + threadIdx.x);

		// Browse tiles of landmarks
		for (int tile = 0; tile < d_max_landmarks; tile += blockDim.x) {
			findWinnerTile4(tile, sample, &bestMatching, &bestDistance);

			__syncthreads();
		}

		/*
		fprintf(stderr, "findWinnersCuda.cu: findWinner4 %d write indices %d %d %f %f\n",
			blockIdx.x * blockDim.x + threadIdx.x, bestMatching.x, bestMatching.y, bestDistance.x, bestDistance.y);
		*/

		// Write indices & distances
		d_indices[blockIdx.x * blockDim.x + threadIdx.x] = bestMatching;
		d_distances[blockIdx.x * blockDim.x + threadIdx.x] = bestDistance;
	}

	// fprintf(stderr, "findWinnersCuda.cu: findWinner4 %d completed\n", blockIdx.x * blockDim.x + threadIdx.x);
}

extern "C" {

int fwk_regsPerThread = 0;
int fwk_threadsPerBlock = 0;
int fwk_blocksPerGrid = 0;
int fwk_sharedMemSize = 0;


void fwk_allocateLandmarks() {
	float *d_landmarks = 0;
	int max_landmarks;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_landmarks, HIP_SYMBOL("d_max_landmarks"), sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void **) &d_landmarks, max_landmarks * sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_landmarks"), &d_landmarks, sizeof(float *)));
}

void fwk_allocateSamples() {
	float *d_samples = 0;
	int max_samples;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_samples, HIP_SYMBOL("d_max_samples"), sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void **) &d_samples, max_samples * sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_samples"), &d_samples, sizeof(float *)));
}

void fwk_allocateIndices() {
	int *d_indices = 0;
	int max_samples;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_samples, HIP_SYMBOL("d_max_samples"), sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void **) &d_indices, max_samples * sizeof(int2)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_indices"), &d_indices, sizeof(int *)));
}

void fwk_allocateDistances() {
	float *d_distances = 0;
	int max_samples;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_samples, HIP_SYMBOL("d_max_samples"), sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void **) &d_distances, max_samples * sizeof(float2)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_distances"), &d_distances, sizeof(float *)));
}

void fwk_setup(int max_landmarks, int max_samples, int dimension) {
	
	// CAUTION: make sure that REGS_PER_THREAD is up to date with .cubin
	fwk_regsPerThread = REGS_PER_THREAD;
	fwk_threadsPerBlock = THREADS_PER_BLOCK;
	fwk_sharedMemSize = fwk_threadsPerBlock * sizeof(float4);
	
	hipDeviceProp_t deviceProp;

	// Consider the default device only
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	
	fwk_blocksPerGrid = max_samples / fwk_threadsPerBlock;
	fwk_blocksPerGrid = (fwk_blocksPerGrid * fwk_threadsPerBlock < max_samples) ?
		fwk_blocksPerGrid + 1 : fwk_blocksPerGrid;
	fwk_blocksPerGrid = (fwk_blocksPerGrid > deviceProp.maxGridSize[0]) ?
		deviceProp.maxGridSize[0] : fwk_blocksPerGrid;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_max_landmarks"), &max_landmarks, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_max_samples"), &max_samples, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_dimension"), &dimension, sizeof(int)));

	/*
	max_landmarks = -1;
	max_samples = -1;
	dimension = -1;

	hipMemcpyFromSymbol(&max_landmarks, HIP_SYMBOL("d_max_landmarks"), sizeof(int));
	hipMemcpyFromSymbol(&max_samples, HIP_SYMBOL("d_max_samples"), sizeof(int));
	hipMemcpyFromSymbol(&dimension, HIP_SYMBOL("d_dimension"), sizeof(int));

	fprintf(stderr, "findWinnersCuda.cu: max_landmarks read %d\n", max_landmarks);
	fprintf(stderr, "findWinnersCuda.cu: max_samples read %d\n", max_samples);
	fprintf(stderr, "findWinnersCuda.cu: dimension read %d\n", dimension);
	*/

	fwk_allocateLandmarks();
	fwk_allocateSamples();
	fwk_allocateIndices();
	fwk_allocateDistances();
}


void fwk_setLandmarks(float *h_landmarks, int from, int to) {
	float *d_landmarks;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_landmarks, HIP_SYMBOL("d_landmarks"), sizeof(float *)));

	CUDA_SAFE_CALL(hipMemcpy(d_landmarks + from, h_landmarks, 
		to * sizeof(float4), hipMemcpyHostToDevice));
}

void fwk_setSamples(float *h_samples, int from, int to) {
	float *d_samples;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_samples, HIP_SYMBOL("d_samples"), sizeof(float *)));

	CUDA_SAFE_CALL(hipMemcpy(d_samples + from, h_samples, 
		to * sizeof(float4), hipMemcpyHostToDevice));
}

void fwk_getLandmarks(float *h_landmarks, int from, int to) {
	float *d_landmarks;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_landmarks, HIP_SYMBOL("d_landmarks"), sizeof(float *)));

	CUDA_SAFE_CALL(hipMemcpy(h_landmarks, d_landmarks + from,
		to * sizeof(float4), hipMemcpyDeviceToHost));
}

void fwk_getSamples(float *h_samples, int from, int to) {
	float *d_samples;
	int dimension;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_samples, HIP_SYMBOL("d_samples"), sizeof(float *)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&dimension, HIP_SYMBOL("d_dimension"), sizeof(int)));

	CUDA_SAFE_CALL(hipMemcpy(h_samples, d_samples + from, 
		to * sizeof(float4), hipMemcpyDeviceToHost));
}

void fwk_getIndices(int *h_indices, int from, int to) {
	int *d_indices;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_indices, HIP_SYMBOL("d_indices"), sizeof(int *)));

	CUDA_SAFE_CALL(hipMemcpy(h_indices, d_indices + from,
		to * sizeof(int2), hipMemcpyDeviceToHost));
}

void fwk_getDistances(float *h_distances, int from, int to) {
	float *d_distances;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_distances, HIP_SYMBOL("d_distances"), sizeof(float *)));

	CUDA_SAFE_CALL(hipMemcpy(h_distances, d_distances + from,
		to * sizeof(float2), hipMemcpyDeviceToHost));
}

void fwk_findWinners() {
	int max_samples, max_landmarks;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_landmarks, HIP_SYMBOL("d_max_landmarks"), sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&max_samples, HIP_SYMBOL("d_max_samples"), sizeof(int)));

	dim3 block(fwk_threadsPerBlock, 1, 1);
	dim3 grid(fwk_blocksPerGrid, 1, 1);

	/*
	fprintf(stderr, "findWinnersCuda.cu: grid {%d, %d, %d}, block {%d, %d, %d}, sharedMemSize %d\n",
		grid.x, grid.y, grid.z,
		block.x, block.y, block.z,
		fwk_sharedMemSize);
	*/

	findWinner4<<<grid, block, fwk_sharedMemSize>>>();

	// check if kernel invocation generated an error
    CUT_CHECK_ERROR("Kernel execution failed");
}

void fwk_threadSyncronize() {
	CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void fwk_releaseLandmarks() {
	float *d_landmarks;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_landmarks, HIP_SYMBOL("d_landmarks"), sizeof(float *)));
	CUDA_SAFE_CALL(hipFree(d_landmarks));

	d_landmarks = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_landmarks"), &d_landmarks, sizeof(float *)));
}

void fwk_releaseSamples() {
	float *d_samples;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_samples, HIP_SYMBOL("d_samples"), sizeof(float *)));
	CUDA_SAFE_CALL(hipFree(d_samples));

	d_samples = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_samples"), &d_samples, sizeof(float *)));
}

void fwk_releaseIndices() {
	int *d_indices;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_indices, HIP_SYMBOL("d_indices"), sizeof(int *)));
	CUDA_SAFE_CALL(hipFree(d_indices));

	d_indices = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_indices"), &d_indices, sizeof(int *)));
}

void fwk_releaseDistances() {
	float *d_distances;

	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&d_distances, HIP_SYMBOL("d_distances"), sizeof(float *)));
	CUDA_SAFE_CALL(hipFree(d_distances));

	d_distances = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_distances"), &d_distances, sizeof(float *)));
}

void fwk_release() {
	int max_landmarks = 0;
	int max_samples = 0;
	int dimension = 0;

	fwk_releaseLandmarks();
	fwk_releaseSamples();
	fwk_releaseIndices();
	fwk_releaseDistances();

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_max_landmarks"), &max_landmarks, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_max_samples"), &max_samples, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_dimension"), &dimension, sizeof(int)));
}


} // extern "C"

